﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

static const long BLOCKS = 256;
static const long THREAD_X_BLOCK = 256;
static const long ITER_X_THREAD = 10000;

__global__
void piMC(long long* blockCounter, unsigned long long seed) { // blockCounter debe tener un contador por cada bloque

	// Debe haber un contador por cada hilo en el bloque (compartido  en el bloque)
	__shared__ long long threadCounter[THREAD_X_BLOCK];

	// ID de la thread
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	// Inicializa el RNG
	hiprandState_t rng;
	hiprand_init(seed, id, 0, &rng);

	// Inicializa el contador
	threadCounter[threadIdx.x] = 0;

	// Calcula los puntos dentro del círculo
	for (int i = 0; i < ITER_X_THREAD; i++) {
		float x = hiprand_uniform(&rng);
		float y = hiprand_uniform(&rng);
		if (x * x + y * y <= 1.0) {
			threadCounter[threadIdx.x] += 1;
		}
	}

	// La primera thread en cada bloque suma los contadores individuales en el de bloque
	if (threadIdx.x == 0) {
		// Inicializa el contador de este bloque
		blockCounter[blockIdx.x] = 0;
		// Suma los contadores de thread en el de bloque
		for (int i = 0; i < THREAD_X_BLOCK; i++) {
			blockCounter[blockIdx.x] += threadCounter[i];
		}
	}
}

int main(void) {
	// Crea el buffer para los contadores de bloque en el host
	long long* blockCounter = (long long*)malloc(sizeof(long long) * BLOCKS);

	// Crea el buffer para los contadores de bloque en la GPU
	long long* gpuBlockCounter;
	hipMalloc(&gpuBlockCounter, sizeof(long long) * BLOCKS);

	// Ejecuta la kernel
	unsigned long long seed = (unsigned long long) time(NULL);
	piMC <<< BLOCKS, THREAD_X_BLOCK >>> (gpuBlockCounter, seed);

	// Recupera el resultado desde la GPU y lo pone en el buffer del host
	hipMemcpy(blockCounter, gpuBlockCounter, sizeof(long long) * BLOCKS, hipMemcpyDeviceToHost);

	// Suma los contadores y calcula PI
	unsigned long long total = 0;
	for (int i = 0; i < BLOCKS; i++) {
		total += blockCounter[i];
	}
	unsigned long long iters = BLOCKS * THREAD_X_BLOCK * ITER_X_THREAD;
	printf("Aproximado con %lld iteraciones\n", iters);
	printf("%lld puntos dentro del círculo\n", total);
	printf("PI= %f\n", 4.0 * ( (double)total / (double)iters ) );

	// Libera los recursos
	hipFree(gpuBlockCounter);
	free(blockCounter);

	return 0;
}
